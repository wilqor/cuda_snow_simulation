
#include <hip/hip_runtime.h>
 extern "C"
 __global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        // sum[i] = a[i] + b[i];
        sum[i] = sinf(a[i] + b[i]);
    }
}

 extern "C"
 __global__ void dalibomba(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        // sum[i] = a[i] + b[i];
        sum[i] = sinf(a[i] + b[i]);
    }
}
