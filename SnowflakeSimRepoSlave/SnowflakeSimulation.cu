
#include <hip/hip_runtime.h>
 extern "C"
 __global__ void prepare(float *positions, int *usageIndexes, int count, int iterations,
                        float *random, float minScale, float maxScale,
                        float minX, float maxX, float minY)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x, jump = gridDim.x * blockDim.x, i,
        row = iterations * 2 + 1;
    float x, y, scale;
    // for each snowflake
    for (i = id; i < count; i += jump)
    {
        // reset usageIndex
        usageIndexes[i] = 0;
        scale = random[i] * (maxScale - minScale) + minScale;
        // starting positions
        x = random[i + 1] * (maxX - minX) + minX;
        y = minY;
        // store in positions
        positions[i * row + 0] = scale;
        positions[i * row + 1] = x;
        positions[i * row + 2] = y;
    }
}

 extern "C"
 __global__ void calculate(float *positions, int *usageIndexes, int count, int iterations,
                            float wind, float angle, float gravity, float maxX, float minX, float maxY)
{
     int id = blockIdx.x * blockDim.x + threadIdx.x, jump = gridDim.x * blockDim.x, i, j, usageIndex,
            row = iterations * 2 + 1;
     float x, y, windX, windY, scale, sin, cos, prevX, prevY;
     // trigonometrics used for wind force
     sincosf(angle, &sin, &cos);
     windX = wind * sin;
     windY = wind * cos;
     // for each snowflake
     for (i = id; i < count; i += jump)
     {
        scale = positions[i * row + 0];
        usageIndex = usageIndexes[i];
        prevX = positions[i * row + 1];
        prevY = positions[i * row + 2];
        // starting from index 2, as <0, 2> is to be prepared earlier
        for (j = 3; j < iterations * 2 + 1; j += 2)
        {
            x = prevX + windX;
            if (x < minX)
            {
                x = maxX + x - minX;
            }
            else if (x > maxX)
            {
                x = minX + x - maxX;
            }
            y = prevY + gravity * scale + windY;
            if (y > maxY && usageIndex == 0)
            {
                usageIndex = j;
                usageIndexes[i] = usageIndex;
            }
            positions[i * row + j] = x;
            positions[i * row + j + 1] = y;
            prevX = x;
            prevY = y;
        }
     }
}
