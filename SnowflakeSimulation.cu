
#include <hip/hip_runtime.h>
 extern "C"
 __global__ void prepare(float *positions, int *usageIndexes, int count, int iterations,
                        float *random, float minScale, float maxScale,
                        float minX, float maxX, float minY)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x, jump = gridDim.x * blockDim.x, i,
        row = iterations * 2 + 1;
    float x, y, scale;
    // for each snowflake
    for (i = id; i < count; i += jump)
    {
        // reset usageIndex
        usageIndexes[i] = 0;
        scale = random[i] * (maxScale - minScale) + minScale;
        // starting positions
        x = random[i + 1] * (maxX - minX) + minX;
        y = minY;
        // store in positions
        positions[i * row + 0] = scale;
        positions[i * row + 1] = x;
        positions[i * row + 2] = y;
    }
}

 extern "C"
 __global__ void calculate(float *positions, int *usageIndexes, int count, int iterations,
                            float wind, float angle, float gravity, float maxX, float minX, float xMargin, float maxY)
{
     int id = blockIdx.x * blockDim.x + threadIdx.x, jump = gridDim.x * blockDim.x, i, j, usageIndex,
            row = iterations * 2 + 1;
     float x, y, windX, windY, scale, sin, cos;
     // trigonometrics used for wind force
     sincosf(angle, &sin, &cos);
     windX = wind * sin;
     windY = wind * cos;
     // for each snowflake
     for (i = id; i < count; i += jump)
     {
        scale = positions[i * row + 0];
        usageIndex = usageIndexes[i];
        // starting from index 2, as <0, 2> is to be prepared earlier
        for (j = 3; j < iterations * 2 + 1; j += 2)
        {
            x = positions[i * row + j - 2] + windX;
            if (x < minX)
            {
                x = maxX - xMargin + x;
            }
            else if (x > maxX)
            {
                x = minX + xMargin + (x - maxX);
            }
            y = positions[i * row + j - 1] + gravity * scale + windY;
            if (y > maxY && usageIndex == 0)
            {
                usageIndex = j;
                usageIndexes[i] = usageIndex;
            }
            positions[i * row + j] = x;
            positions[i * row + j + 1] = y;
        }
     }
}
